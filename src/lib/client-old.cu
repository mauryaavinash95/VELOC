#include "hip/hip_runtime.h"
#include "client.hpp"
#include "include/veloc.h"
#include "common/file_util.hpp"
#include <vector>
#include <fstream>
#include <stdexcept>
#include <regex>
#include <unistd.h>
#include <limits.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define __DEBUG
#include "common/debug.hpp"

static bool validate_name(const char *name) {
    std::regex e("[a-zA-Z0-9_\\.]+");
    return std::regex_match(name, e);
}

static void launch_backend(const char *cfg_file) {
    char *path = getenv("VELOC_BIN");
    std::string command;
    if (path != NULL)
        command = std::string(path) + "/";
    command += "veloc-backend " + std::string(cfg_file) + " --disable-ec > /dev/null";
    if (system(command.c_str()) != 0)
        FATAL("cannot launch active backend for async mode, error: " << strerror(errno));
}

veloc_client_t::veloc_client_t(unsigned int id, const char *cfg_file) :
    cfg(cfg_file), collective(false), rank(id) {
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    DBG("VELOC initialized");
}

veloc_client_t::veloc_client_t(MPI_Comm c, const char *cfg_file) :
    cfg(cfg_file), comm(c), collective(true) {
    MPI_Comm_rank(comm, &rank);
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg, comm, true);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    DBG("VELOC initialized");
}

veloc_client_t::~veloc_client_t() {
    delete queue;
    delete modules;
    DBG("VELOC finalized");
}

bool veloc_client_t::mem_protect(int id, void *ptr, size_t count, size_t base_size, unsigned int flags=NULL, release=NULL ) {
    // mem_regions[id] = std::make_pair(ptr, base_size * count);
    mem_regions[id] = std::make_tuple(ptr, base_size * count, flags, release);
    return true;
}

bool veloc_client_t::mem_unprotect(int id) {
    return mem_regions.erase(id) > 0;
}

bool veloc_client_t::checkpoint_wait() {
    if (cfg.is_sync())
	return true;
    if (checkpoint_in_progress) {
	ERROR("need to finalize local checkpoint first by calling checkpoint_end()");
	return false;
    }
    return queue->wait_completion() == VELOC_SUCCESS;
}

bool veloc_client_t::checkpoint_begin(const char *name, int version) {
    TIMER_START(io_timer_ckpt_begin);
    if (checkpoint_in_progress) {
	ERROR("nested checkpoints not yet supported");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return false;
    }

    DBG("called checkpoint_begin");
    current_ckpt = command_t(rank, command_t::CHECKPOINT, version, name);
    checkpoint_in_progress = true;
    TIMER_STOP(io_timer_ckpt_begin, " --- CKPT BEGIN TIME --- ");
    return true;
}

bool veloc_client_t::checkpoint_mem(int mode, std::set<int> &ids) {
    TIMER_START(io_timer_ckpt_mem);
    DBG("Starting checkpoint_mem");
    if (!checkpoint_in_progress) {
	ERROR("must call checkpoint_begin() first");
	return false;
    }
    regions_t ckpt_regions;
    if (mode == VELOC_CKPT_ALL)
        ckpt_regions = mem_regions;
    else if (mode == VELOC_CKPT_SOME) {
        for (auto it = ids.begin(); it != ids.end(); it++) {
            auto found = mem_regions.find(*it);
            if (found != mem_regions.end())
                ckpt_regions.insert(*found);
        }
    } else if (mode == VELOC_CKPT_REST) {
        ckpt_regions = mem_regions;
        for (auto it = ids.begin(); it != ids.end(); it++)
            ckpt_regions.erase(*it);
    }
    if (ckpt_regions.size() == 0) {
	ERROR("empty selection of memory regions to checkpoint, please check protection and/or selective checkpointing primitives");
	return false;
    }

    std::ofstream f;
    f.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    try {
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
	size_t regions_size = ckpt_regions.size();
    f.write((char *)&regions_size, sizeof(size_t));
    hipPointerAttribute_t attributes;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    std::vector<char *> temp_ptrs;
	for (auto &e : ckpt_regions) {
	    f.write((char *)&(e.first), sizeof(int));
        f.write((char *)&(e.second.second), sizeof(size_t));
        hipPointerGetAttributes (&attributes, e.second.first);
        // attributes.type can be one of cudaMemoryTypeUnregistered (unpinned), 
        // hipMemoryTypeHost (pinned), hipMemoryTypeDevice (on-GPU), or hipMemoryTypeManaged (managed)
        if(attributes.type==hipMemoryTypeDevice || attributes.type==hipMemoryTypeManaged) {
            // Copy from device to host first, and then ckpt.
            char *temp;
            char *gpu_var = (char *)e.second.first;
            hipEventRecord(start);
            hipHostMalloc((void**)&temp, e.second.second);
            hipMemcpy(temp, gpu_var, e.second.second, hipMemcpyDeviceToHost);
            temp_ptrs.push_back(temp);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            DBG("[CUDA TIME] Transferring " << e.first << " took " << milliseconds << " ms" );
            ckpt_regions[e.first] = std::make_pair(temp, e.second.second);
        }
    }
    for (auto &e : ckpt_regions)
        f.write((char *)e.second.first, e.second.second);
    for (char *t : temp_ptrs)
        hipHostFree(t);
    TIMER_STOP(io_timer_ckpt_mem, " --- CKPT MEM TIME --- ");
    } catch (std::ofstream::failure &f) {
	ERROR("cannot write to checkpoint file: " << current_ckpt << ", reason: " << f.what());
	return false;
    }
    return true;
}

bool veloc_client_t::checkpoint_end(bool /*success*/) {
    TIMER_START(io_timer_ckpt_end);
    checkpoint_in_progress = false;
    if (cfg.is_sync()) {
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return modules->notify_command(current_ckpt) == VELOC_SUCCESS;
    }
    else {
        queue->enqueue(current_ckpt);
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return true;
    }
}

int veloc_client_t::run_blocking(const command_t &cmd) {
    if (cfg.is_sync())
	return modules->notify_command(cmd);
    else {
	queue->enqueue(cmd);
	return queue->wait_completion();
    }
}

int veloc_client_t::restart_test(const char *name, int needed_version) {
    if (!validate_name(name) || needed_version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }
    int version = run_blocking(command_t(rank, command_t::TEST, needed_version, name));
    DBG(name << ": latest version = " << version);
    if (collective) {
	int min_version;
	MPI_Allreduce(&version, &min_version, 1, MPI_INT, MPI_MIN, comm);
	return min_version;
    } else
	return version;
}

std::string veloc_client_t::route_file(const char *original) {
    char abs_path[PATH_MAX + 1];
    if (original[0] != '/' && getcwd(abs_path, PATH_MAX) != NULL)
	current_ckpt.assign_path(current_ckpt.original, std::string(abs_path) + "/" + std::string(original));
    else
	current_ckpt.assign_path(current_ckpt.original, std::string(original));
    return current_ckpt.filename(cfg.get("scratch"));
}

bool veloc_client_t::restart_begin(const char *name, int version) {
    if (checkpoint_in_progress) {
	INFO("cannot restart while checkpoint in progress");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }

    int result, end_result;
    current_ckpt = command_t(rank, command_t::RESTART, version, name);
    result = run_blocking(current_ckpt);
    if (collective)
	MPI_Allreduce(&result, &end_result, 1, MPI_INT, MPI_LOR, comm);
    else
	end_result = result;
    if (end_result == VELOC_SUCCESS) {
        header_size = 0;
	return true;
    } else
	return false;
}

bool veloc_client_t::read_header() {
    region_info.clear();
    try {
	std::ifstream f;
        size_t expected_size = 0;

	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	size_t no_regions, region_size;
	int id;
	f.read((char *)&no_regions, sizeof(size_t));
	for (unsigned int i = 0; i < no_regions; i++) {
	    f.read((char *)&id, sizeof(int));
	    f.read((char *)&region_size, sizeof(size_t));
	    region_info.insert(std::make_pair(id, region_size));
            expected_size += region_size;
	}
	header_size = f.tellg();
        f.seekg(0, f.end);
        size_t file_size = (size_t)f.tellg() - header_size;
        if (file_size != expected_size)
            throw std::ifstream::failure("file size " + std::to_string(file_size) + " does not match expected size " + std::to_string(expected_size));
    } catch (std::ifstream::failure &e) {
	ERROR("cannot validate header for checkpoint " << current_ckpt << ", reason: " << e.what());
	header_size = 0;
	return false;
    }
    return true;
}

size_t veloc_client_t::recover_size(int id) {
    if (header_size == 0)
        read_header();
    auto it = region_info.find(id);
    if (it == region_info.end())
	return 0;
    else
	return it->second;
}

bool veloc_client_t::recover_mem(int mode, std::set<int> &ids) {
    if (header_size == 0 && !read_header()) {
	ERROR("cannot recover in memory mode if header unavailable or corrupted");
	return false;
    }
    try {
	std::ifstream f;
	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	f.seekg(header_size);
	for (auto &e : region_info) {
	    bool found = ids.find(e.first) != ids.end();
	    if ((mode == VELOC_RECOVER_SOME && !found) || (mode == VELOC_RECOVER_REST && found)) {
		f.seekg(e.second, std::ifstream::cur);
		continue;
	    }
	    if (mem_regions.find(e.first) == mem_regions.end()) {
		ERROR("no protected memory region defined for id " << e.first);
		return false;
	    }
	    if (mem_regions[e.first].second < e.second) {
		ERROR("protected memory region " << e.first << " is too small ("
		      << mem_regions[e.first].second << ") to hold required size ("
		      << e.second << ")");
		return false;
	    }
	    f.read((char *)mem_regions[e.first].first, e.second);
	}
    } catch (std::ifstream::failure &e) {
	ERROR("cannot read checkpoint file " << current_ckpt << ", reason: " << e.what());
	return false;
    }
    return true;
}

bool veloc_client_t::restart_end(bool /*success*/) {
    return true;
}
