#include "hip/hip_runtime.h"
#include "client.hpp"
#include "include/veloc.h"
#include "common/file_util.hpp"
#include <vector>
#include <fstream>
#include <stdexcept>
#include <regex>
#include <unistd.h>
#include <limits.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define __DEBUG
#include "common/debug.hpp"

static bool validate_name(const char *name) {
    std::regex e("[a-zA-Z0-9_\\.]+");
    return std::regex_match(name, e);
}

static void launch_backend(const char *cfg_file) {
    char *path = getenv("VELOC_BIN");
    std::string command;
    if (path != NULL)
        command = std::string(path) + "/";
    command += "veloc-backend " + std::string(cfg_file) + " --disable-ec > /dev/null";
    if (system(command.c_str()) != 0)
        FATAL("cannot launch active backend for async mode, error: " << strerror(errno));
}

veloc_client_t::veloc_client_t(unsigned int id, const char *cfg_file) :
    cfg(cfg_file), collective(false), rank(id) {
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    hipStreamCreate(&veloc_stream);
    gpu_memcpy_thread = std::thread([&] { checkpoint_gpu_mem(); });
    gpu_memcpy_thread.detach();
    DBG("VELOC initialized");
}

veloc_client_t::veloc_client_t(MPI_Comm c, const char *cfg_file) :
    cfg(cfg_file), comm(c), collective(true) {
    MPI_Comm_rank(comm, &rank);
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg, comm, true);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    hipStreamCreate(&veloc_stream);
    gpu_memcpy_thread = std::thread([&] { checkpoint_gpu_mem(); });
    gpu_memcpy_thread.detach();
    DBG("VELOC initialized");
}

veloc_client_t::~veloc_client_t() {
    delete queue;
    delete modules;
    hipStreamDestroy(veloc_stream);
    DBG("VELOC finalized");
}

bool veloc_client_t::mem_protect(int id, void *ptr, size_t count, size_t base_size, unsigned int flags=0, release release_routine=NULL ) {
    // mem_regions[id] = std::make_pair(ptr, base_size * count);
    mem_regions[id] = std::make_tuple(ptr, base_size * count, flags, release_routine);
    return true;
}

bool veloc_client_t::mem_unprotect(int id) {
    return mem_regions.erase(id) > 0;
}

bool veloc_client_t::checkpoint_wait() {
    if (cfg.is_sync())
	return true;
    if (checkpoint_in_progress) {
	ERROR("need to finalize local checkpoint first by calling checkpoint_end()");
	return false;
    }
    return queue->wait_completion() == VELOC_SUCCESS;
}

bool veloc_client_t::checkpoint_begin(const char *name, int version) {
    TIMER_START(io_timer_ckpt_begin);
    if (checkpoint_in_progress) {
	ERROR("nested checkpoints not yet supported");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return false;
    }

    DBG("called checkpoint_begin");
    current_ckpt = command_t(rank, command_t::CHECKPOINT, version, name);
    checkpoint_in_progress = true;
    ckpt_check_done = false;
    TIMER_STOP(io_timer_ckpt_begin, " --- CKPT BEGIN TIME --- ");
    return true;
}

bool veloc_client_t::checkpoint_gpu_mem() {
    void *ptr; size_t sz; 
    do {
        std::unique_lock<std::mutex> lock(gpu_memcpy_mutex);
        while (gpu_memcpy_regions.empty()){
            gpu_memcpy_cv.wait(lock, [&](){ return !gpu_memcpy_regions.empty(); }); 
        }
        while (!gpu_memcpy_regions.empty()) {
            std::unique_lock<std::mutex> ul(gpu_memcpy_done_mutex);
            gpu_memcpy_done = false;
            auto e = gpu_memcpy_regions.begin();
            char *temp;
            ptr = std::get<0>(e->second);
            sz = std::get<1>(e->second);
            hipHostMalloc((void**)&temp, sz);
            hipMemcpyAsync(temp, ptr, sz, hipMemcpyDeviceToHost, veloc_stream);
            temp_host_ptrs.push_back(temp);
            async_gpu_regions[e->first] = e->second;
            gpu_memcpy_regions.erase(e);
        }
    } while (!ckpt_check_done);
    hipStreamSynchronize(veloc_stream);
    gpu_memcpy_done = true;
    gpu_memcpy_done_cv.notify_one();
    return true;
}

bool veloc_client_t::checkpoint_mem(int mode, std::set<int> &ids) {
    TIMER_START(io_timer_ckpt_mem);
    DBG("Starting checkpoint_mem");
    if (!checkpoint_in_progress) {
        ERROR("must call checkpoint_begin() first");
        return false;
    }
    regions_t ckpt_regions;
    if (mode == VELOC_CKPT_ALL)
        ckpt_regions = mem_regions;
    else if (mode == VELOC_CKPT_SOME) {
        for (auto it = ids.begin(); it != ids.end(); it++) {
            auto found = mem_regions.find(*it);
            if (found != mem_regions.end())
                ckpt_regions.insert(*found);
        }
    } else if (mode == VELOC_CKPT_REST) {
        ckpt_regions = mem_regions;
        for (auto it = ids.begin(); it != ids.end(); it++)
            ckpt_regions.erase(*it);
    }
    if (ckpt_regions.size() == 0) {
        ERROR("empty selection of memory regions to checkpoint, please check protection and/or selective checkpointing primitives");
        return false;
    }

    float gpu_cache = std::stof(current_ckpt.filename(cfg.get("gpu_cache_size")));
    float rem_gpu_cache = (1<<30)*gpu_cache;
    DBG("Allowed " << rem_gpu_cache << " GPU cache size.");

    async_gpu_regions.clear();
    hipPointerAttribute_t attributes;
    void *ptr; size_t sz; unsigned int flags=0; release release_routine=NULL;    
    size_t free_gpu_mem, total_gpu_mem;

    // In this loop, we check for the GPU memory transfer conditions,
    // If gpu_cache_space is available, we either make a copy or start
    // async ckpt of the original dev pointer - We add in gpu_memory_regions 
    // and notify the thread 
    // Else, if gpu_cache_space is unavailable, we perform blocking transfer from
    // the device to host memory.
    for (auto &e : ckpt_regions) {
        ptr = std::get<0>(e.second);
        sz = std::get<1>(e.second);
        flags = std::get<2>(e.second);
        release_routine = std::get<3>(e.second); 
        hipPointerGetAttributes (&attributes, ptr);
        if(attributes.type==hipMemoryTypeDevice) {
            hipMemGetInfo(&free_gpu_mem, &total_gpu_mem);
            if(free_gpu_mem >= sz && rem_gpu_cache >= sz) {
                char *new_ptr = (char *)ptr;
                if(flags == DEFAULT) {
                    hipMalloc((void**)&new_ptr, sz);
                    hipMemcpy(new_ptr, ptr, sz, hipMemcpyDeviceToDevice);
                    temp_dev_ptrs.push_back(new_ptr);
                }
                rem_gpu_cache -= sz;
                std::get<0>(e.second) = new_ptr;
                std::unique_lock<std::mutex> lock(gpu_memcpy_mutex);
                gpu_memcpy_regions.insert(e);
                gpu_memcpy_cv.notify_one();
            } else {
                char *temp;
                hipHostMalloc((void**)&temp, sz);
                hipMemcpy(temp, ptr, sz, hipMemcpyDeviceToHost);
                temp_host_ptrs.push_back(temp);
                std::get<0>(ckpt_regions[e.first]) = temp;    
            }
        }
    }
    ckpt_check_done = true;

    // Wait for the GPU async ckpt method to flush to file first.
    std::unique_lock<std::mutex> lock(gpu_memcpy_done_mutex);
    while(!gpu_memcpy_done) {
        gpu_memcpy_done_cv.wait(lock, [&](){ return gpu_memcpy_done; } );
    }

    // Edit the regions to be checkpointed from the GPU async from the ckpt_regions list
    for (auto &e : async_gpu_regions)
        std::get<0>(ckpt_regions[e.first]) = std::get<0>(e.second);

    TIMER_START(io_timer_ckpt_host_mem);
    bool ret = mem_write(ckpt_regions);
    TIMER_STOP(io_timer_ckpt_host_mem, " --- CKPT HOST MEM TIME --- ");

    for (char *t : temp_host_ptrs)
        hipHostFree(t);
    
    for (char *t : temp_dev_ptrs)
        hipFree(t);

    temp_host_ptrs.clear();
    temp_dev_ptrs.clear();

    TIMER_STOP(io_timer_ckpt_mem, " --- CKPT MEM TIME --- ");
    return ret;
}

bool veloc_client_t::mem_write(regions_t ckpt_regions) {
    std::ofstream f;
    f.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    try {
        f.open(current_ckpt.filename(cfg.get("scratch")), std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
        size_t regions_size = ckpt_regions.size();
        f.write((char *)&regions_size, sizeof(size_t));
        for (auto &e : ckpt_regions) {
            f.write((char *)&(e.first), sizeof(int));
            f.write((char *)&(std::get<1>(e.second)), sizeof(size_t));
        }  
        
        for (auto &e : ckpt_regions)
            f.write((char *)std::get<0>(e.second), std::get<1>(e.second));
    } catch (std::ofstream::failure &f) {
        ERROR("cannot write to checkpoint file: " << current_ckpt << ", reason: " << f.what());
        return false;
    }
    return true;
}

bool veloc_client_t::checkpoint_end(bool /*success*/) {
    TIMER_START(io_timer_ckpt_end);
    checkpoint_in_progress = false;
    if (cfg.is_sync()) {
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return modules->notify_command(current_ckpt) == VELOC_SUCCESS;
    }
    else {
        queue->enqueue(current_ckpt);
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return true;
    }
}

int veloc_client_t::run_blocking(const command_t &cmd) {
    if (cfg.is_sync())
	return modules->notify_command(cmd);
    else {
	queue->enqueue(cmd);
	return queue->wait_completion();
    }
}

int veloc_client_t::restart_test(const char *name, int needed_version) {
    if (!validate_name(name) || needed_version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }
    int version = run_blocking(command_t(rank, command_t::TEST, needed_version, name));
    DBG(name << ": latest version = " << version);
    if (collective) {
	int min_version;
	MPI_Allreduce(&version, &min_version, 1, MPI_INT, MPI_MIN, comm);
	return min_version;
    } else
	return version;
}

std::string veloc_client_t::route_file(const char *original) {
    char abs_path[PATH_MAX + 1];
    if (original[0] != '/' && getcwd(abs_path, PATH_MAX) != NULL)
	current_ckpt.assign_path(current_ckpt.original, std::string(abs_path) + "/" + std::string(original));
    else
	current_ckpt.assign_path(current_ckpt.original, std::string(original));
    return current_ckpt.filename(cfg.get("scratch"));
}

bool veloc_client_t::restart_begin(const char *name, int version) {
    if (checkpoint_in_progress) {
	INFO("cannot restart while checkpoint in progress");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }

    int result, end_result;
    current_ckpt = command_t(rank, command_t::RESTART, version, name);
    result = run_blocking(current_ckpt);
    if (collective)
	MPI_Allreduce(&result, &end_result, 1, MPI_INT, MPI_LOR, comm);
    else
	end_result = result;
    if (end_result == VELOC_SUCCESS) {
        header_size = 0;
	return true;
    } else
	return false;
}

bool veloc_client_t::read_header() {
    region_info.clear();
    try {
	std::ifstream f;
        size_t expected_size = 0;

	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	size_t no_regions, region_size;
	int id;
	f.read((char *)&no_regions, sizeof(size_t));
	for (unsigned int i = 0; i < no_regions; i++) {
	    f.read((char *)&id, sizeof(int));
	    f.read((char *)&region_size, sizeof(size_t));
	    region_info.insert(std::make_pair(id, region_size));
            expected_size += region_size;
	}
	header_size = f.tellg();
        f.seekg(0, f.end);
        size_t file_size = (size_t)f.tellg() - header_size;
        if (file_size != expected_size)
            throw std::ifstream::failure("file size " + std::to_string(file_size) + " does not match expected size " + std::to_string(expected_size));
    } catch (std::ifstream::failure &e) {
	ERROR("cannot validate header for checkpoint " << current_ckpt << ", reason: " << e.what());
	header_size = 0;
	return false;
    }
    return true;
}

size_t veloc_client_t::recover_size(int id) {
    if (header_size == 0)
        read_header();
    auto it = region_info.find(id);
    if (it == region_info.end())
	return 0;
    else
	return it->second;
}

bool veloc_client_t::recover_mem(int mode, std::set<int> &ids) {
    if (header_size == 0 && !read_header()) {
	ERROR("cannot recover in memory mode if header unavailable or corrupted");
	return false;
    }
    try {
	std::ifstream f;
	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	f.seekg(header_size);
	for (auto &e : region_info) {
	    bool found = ids.find(e.first) != ids.end();
	    if ((mode == VELOC_RECOVER_SOME && !found) || (mode == VELOC_RECOVER_REST && found)) {
		f.seekg(e.second, std::ifstream::cur);
		continue;
	    }
	    if (mem_regions.find(e.first) == mem_regions.end()) {
		ERROR("no protected memory region defined for id " << e.first);
		return false;
	    }
	    if (std::get<1>(mem_regions[e.first]) < e.second) {
		ERROR("protected memory region " << e.first << " is too small ("
		      << std::get<1>(mem_regions[e.first]) << ") to hold required size ("
		      << e.second << ")");
		return false;
	    }
	    f.read((char *)std::get<0>(mem_regions[e.first]), e.second);
	}
    } catch (std::ifstream::failure &e) {
	ERROR("cannot read checkpoint file " << current_ckpt << ", reason: " << e.what());
	return false;
    }
    return true;
}

bool veloc_client_t::restart_end(bool /*success*/) {
    return true;
}
