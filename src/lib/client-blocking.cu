#include "hip/hip_runtime.h"
#include "client.hpp"
#include "include/veloc.h"
#include "common/file_util.hpp"
#include <vector>
#include <fstream>
#include <stdexcept>
#include <regex>
#include <unistd.h>
#include <limits.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define __DEBUG
#include "common/debug.hpp"

static bool validate_name(const char *name) {
    std::regex e("[a-zA-Z0-9_\\.]+");
    return std::regex_match(name, e);
}

static void launch_backend(const char *cfg_file) {
    char *path = getenv("VELOC_BIN");
    std::string command;
    if (path != NULL)
        command = std::string(path) + "/";
    command += "veloc-backend " + std::string(cfg_file) + " --disable-ec > /dev/null";
    if (system(command.c_str()) != 0)
        FATAL("cannot launch active backend for async mode, error: " << strerror(errno));
}

veloc_client_t::veloc_client_t(unsigned int id, const char *cfg_file) :
    cfg(cfg_file), collective(false), rank(id) {
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    DBG("VELOC initialized");
}

veloc_client_t::veloc_client_t(MPI_Comm c, const char *cfg_file) :
    cfg(cfg_file), comm(c), collective(true) {
    MPI_Comm_rank(comm, &rank);
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg, comm, true);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    DBG("VELOC initialized");
}

veloc_client_t::~veloc_client_t() {
    delete queue;
    delete modules;
    DBG("VELOC finalized");
}

bool veloc_client_t::mem_protect(int id, void *ptr, size_t count, size_t base_size, unsigned int flags=0, release release_routine=NULL ) {
    // mem_regions[id] = std::make_pair(ptr, base_size * count);
    mem_regions[id] = std::make_tuple(ptr, base_size * count, flags, release_routine);
    return true;
}

bool veloc_client_t::mem_unprotect(int id) {
    return mem_regions.erase(id) > 0;
}

bool veloc_client_t::checkpoint_wait() {
    if (cfg.is_sync())
	return true;
    if (checkpoint_in_progress) {
	ERROR("need to finalize local checkpoint first by calling checkpoint_end()");
	return false;
    }
    return queue->wait_completion() == VELOC_SUCCESS;
}

bool veloc_client_t::checkpoint_begin(const char *name, int version) {
    TIMER_START(io_timer_ckpt_begin);
    if (checkpoint_in_progress) {
	ERROR("nested checkpoints not yet supported");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return false;
    }

    DBG("called checkpoint_begin");
    current_ckpt = command_t(rank, command_t::CHECKPOINT, version, name);
    checkpoint_in_progress = true;
    TIMER_STOP(io_timer_ckpt_begin, " --- CKPT BEGIN TIME --- ");
    return true;
}

bool veloc_client_t::checkpoint_mem(int mode, std::set<int> &ids) {
    TIMER_START(io_timer_ckpt_mem);
    DBG("Starting checkpoint_mem");
    if (!checkpoint_in_progress) {
	ERROR("must call checkpoint_begin() first");
	return false;
    }
    regions_t ckpt_regions;
    if (mode == VELOC_CKPT_ALL)
        ckpt_regions = mem_regions;
    else if (mode == VELOC_CKPT_SOME) {
        for (auto it = ids.begin(); it != ids.end(); it++) {
            auto found = mem_regions.find(*it);
            if (found != mem_regions.end())
                ckpt_regions.insert(*found);
        }
    } else if (mode == VELOC_CKPT_REST) {
        ckpt_regions = mem_regions;
        for (auto it = ids.begin(); it != ids.end(); it++)
            ckpt_regions.erase(*it);
    }
    if (ckpt_regions.size() == 0) {
	ERROR("empty selection of memory regions to checkpoint, please check protection and/or selective checkpointing primitives");
	return false;
    }

    std::ofstream f;
    f.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    try {
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
	size_t regions_size = ckpt_regions.size();
    f.write((char *)&regions_size, sizeof(size_t));
    hipPointerAttribute_t attributes;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    void *ptr; size_t sz; unsigned int flags=0; release release_routine=NULL;
    std::vector<char *> temp_ptrs;
    size_t free_gpu_mem, total_gpu_mem;
	for (auto &e : ckpt_regions) {
        f.write((char *)&(e.first), sizeof(int));
        ptr = std::get<0>(e.second);
        sz = std::get<1>(e.second);
        flags = std::get<2>(e.second);
        release_routine = std::get<3>(e.second); 
        f.write((char *)&(sz), sizeof(size_t));
        hipPointerGetAttributes (&attributes, ptr);
        // attributes.type can be one of cudaMemoryTypeUnregistered (unpinned), 
        // hipMemoryTypeHost (pinned), hipMemoryTypeDevice (on-GPU), or hipMemoryTypeManaged (managed)
        if(attributes.type==hipMemoryTypeDevice) {
            hipMemGetInfo(&free_gpu_mem, &total_gpu_mem);
            char *new_ptr;
            if(free_gpu_mem >= sz && flags == DEFAULT) {
                    hipMalloc((void**)&new_ptr, sz);
                    hipMemcpy(&new_ptr, &ptr, sz, hipMemcpyDeviceToDevice);
            } else {
                new_ptr = (char *)ptr;
            }
            char *temp;
            hipEventRecord(start);
            hipHostMalloc((void**)&temp, sz);
            hipMemcpy(temp, new_ptr, sz, hipMemcpyDeviceToHost);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            DBG("[CUDA TIME] Transferring " << e.first << " took " << milliseconds << " ms" );
            temp_ptrs.push_back(temp);
            ckpt_regions[e.first] = std::make_tuple(temp, sz, flags, release_routine);
        }
    }
    for (auto &e : ckpt_regions)
        f.write((char *)std::get<0>(e.second), std::get<1>(e.second));
    for (char *t : temp_ptrs)
        hipHostFree(t);
    TIMER_STOP(io_timer_ckpt_mem, " --- CKPT MEM TIME --- ");
    } catch (std::ofstream::failure &f) {
	ERROR("cannot write to checkpoint file: " << current_ckpt << ", reason: " << f.what());
	return false;
    }
    return true;
}

bool veloc_client_t::checkpoint_end(bool /*success*/) {
    TIMER_START(io_timer_ckpt_end);
    checkpoint_in_progress = false;
    if (cfg.is_sync()) {
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return modules->notify_command(current_ckpt) == VELOC_SUCCESS;
    }
    else {
        queue->enqueue(current_ckpt);
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return true;
    }
}

int veloc_client_t::run_blocking(const command_t &cmd) {
    if (cfg.is_sync())
	return modules->notify_command(cmd);
    else {
	queue->enqueue(cmd);
	return queue->wait_completion();
    }
}

int veloc_client_t::restart_test(const char *name, int needed_version) {
    if (!validate_name(name) || needed_version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }
    int version = run_blocking(command_t(rank, command_t::TEST, needed_version, name));
    DBG(name << ": latest version = " << version);
    if (collective) {
	int min_version;
	MPI_Allreduce(&version, &min_version, 1, MPI_INT, MPI_MIN, comm);
	return min_version;
    } else
	return version;
}

std::string veloc_client_t::route_file(const char *original) {
    char abs_path[PATH_MAX + 1];
    if (original[0] != '/' && getcwd(abs_path, PATH_MAX) != NULL)
	current_ckpt.assign_path(current_ckpt.original, std::string(abs_path) + "/" + std::string(original));
    else
	current_ckpt.assign_path(current_ckpt.original, std::string(original));
    return current_ckpt.filename(cfg.get("scratch"));
}

bool veloc_client_t::restart_begin(const char *name, int version) {
    if (checkpoint_in_progress) {
	INFO("cannot restart while checkpoint in progress");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }

    int result, end_result;
    current_ckpt = command_t(rank, command_t::RESTART, version, name);
    result = run_blocking(current_ckpt);
    if (collective)
	MPI_Allreduce(&result, &end_result, 1, MPI_INT, MPI_LOR, comm);
    else
	end_result = result;
    if (end_result == VELOC_SUCCESS) {
        header_size = 0;
	return true;
    } else
	return false;
}

bool veloc_client_t::read_header() {
    region_info.clear();
    try {
	std::ifstream f;
        size_t expected_size = 0;

	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	size_t no_regions, region_size;
	int id;
	f.read((char *)&no_regions, sizeof(size_t));
	for (unsigned int i = 0; i < no_regions; i++) {
	    f.read((char *)&id, sizeof(int));
	    f.read((char *)&region_size, sizeof(size_t));
	    region_info.insert(std::make_pair(id, region_size));
            expected_size += region_size;
	}
	header_size = f.tellg();
        f.seekg(0, f.end);
        size_t file_size = (size_t)f.tellg() - header_size;
        if (file_size != expected_size)
            throw std::ifstream::failure("file size " + std::to_string(file_size) + " does not match expected size " + std::to_string(expected_size));
    } catch (std::ifstream::failure &e) {
	ERROR("cannot validate header for checkpoint " << current_ckpt << ", reason: " << e.what());
	header_size = 0;
	return false;
    }
    return true;
}

size_t veloc_client_t::recover_size(int id) {
    if (header_size == 0)
        read_header();
    auto it = region_info.find(id);
    if (it == region_info.end())
	return 0;
    else
	return it->second;
}

bool veloc_client_t::recover_mem(int mode, std::set<int> &ids) {
    if (header_size == 0 && !read_header()) {
	ERROR("cannot recover in memory mode if header unavailable or corrupted");
	return false;
    }
    try {
	std::ifstream f;
	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	f.seekg(header_size);
	for (auto &e : region_info) {
	    bool found = ids.find(e.first) != ids.end();
	    if ((mode == VELOC_RECOVER_SOME && !found) || (mode == VELOC_RECOVER_REST && found)) {
		f.seekg(e.second, std::ifstream::cur);
		continue;
	    }
	    if (mem_regions.find(e.first) == mem_regions.end()) {
		ERROR("no protected memory region defined for id " << e.first);
		return false;
	    }
	    if (std::get<1>(mem_regions[e.first]) < e.second) {
		ERROR("protected memory region " << e.first << " is too small ("
		      << std::get<1>(mem_regions[e.first]) << ") to hold required size ("
		      << e.second << ")");
		return false;
	    }
	    f.read((char *)std::get<0>(mem_regions[e.first]), e.second);
	}
    } catch (std::ifstream::failure &e) {
	ERROR("cannot read checkpoint file " << current_ckpt << ", reason: " << e.what());
	return false;
    }
    return true;
}

bool veloc_client_t::restart_end(bool /*success*/) {
    return true;
}
