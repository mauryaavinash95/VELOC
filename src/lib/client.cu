#include "hip/hip_runtime.h"
#include "client.hpp"
#include "include/veloc.h"
#include "common/file_util.hpp"
#include <vector>
#include <fstream>
#include <stdexcept>
#include <regex>
#include <unistd.h>
#include <limits.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define __DEBUG
#include "common/debug.hpp"

static bool validate_name(const char *name) {
    std::regex e("[a-zA-Z0-9_\\.]+");
    return std::regex_match(name, e);
}

static void launch_backend(const char *cfg_file) {
    char *path = getenv("VELOC_BIN");
    std::string command;
    if (path != NULL)
        command = std::string(path) + "/";
    command += "veloc-backend " + std::string(cfg_file) + " --disable-ec > /dev/null";
    if (system(command.c_str()) != 0)
        FATAL("cannot launch active backend for async mode, error: " << strerror(errno));
}

veloc_client_t::veloc_client_t(unsigned int id, const char *cfg_file) :
    cfg(cfg_file), collective(false), rank(id) {
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    hipStreamCreate(&veloc_stream);
    gpu_memcpy_thread = std::thread([&] { gpu_to_host_trf(); });
    write_to_file_thread = std::thread([&] { mem_to_file_write(); });
    gpu_memcpy_thread.detach();
    write_to_file_thread.detach();
    DBG("VELOC initialized");
}

veloc_client_t::veloc_client_t(MPI_Comm c, const char *cfg_file) :
    cfg(cfg_file), comm(c), collective(true) {
    MPI_Comm_rank(comm, &rank);
    if (cfg.is_sync()) {
	modules = new module_manager_t();
	modules->add_default_modules(cfg, comm, true);
    } else {
        launch_backend(cfg_file);
	queue = new client_t<command_t>(rank);
    }
    ec_active = run_blocking(command_t(rank, command_t::INIT, 0, "")) > 0;
    hipStreamCreate(&veloc_stream);
    gpu_memcpy_thread = std::thread([&] { gpu_to_host_trf(); });
    write_to_file_thread = std::thread([&] { mem_to_file_write(); });
    gpu_memcpy_thread.detach();
    write_to_file_thread.detach();
    DBG("VELOC initialized");
}

veloc_client_t::~veloc_client_t() {
    delete queue;
    delete modules;
    veloc_client_active = false;
    hipStreamDestroy(veloc_stream);
    DBG("VELOC finalized");
}

bool veloc_client_t::mem_protect(int id, void *ptr, size_t count, size_t base_size, unsigned int flags=0, release release_routine=NULL ) {
    // mem_regions[id] = std::make_pair(ptr, base_size * count);
    mem_regions[id] = std::make_tuple(ptr, base_size * count, flags, release_routine);
    return true;
}

bool veloc_client_t::mem_unprotect(int id) {
    return mem_regions.erase(id) > 0;
}

bool veloc_client_t::checkpoint_wait() {
    if (cfg.is_sync())
	return true;
    if (checkpoint_in_progress) {
	ERROR("need to finalize local checkpoint first by calling checkpoint_end()");
	return false;
    }
    return queue->wait_completion() == VELOC_SUCCESS;
}

bool veloc_client_t::checkpoint_begin(const char *name, int version) {
    TIMER_START(io_timer_ckpt_begin);
    if (checkpoint_in_progress) {
	ERROR("nested checkpoints not yet supported");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return false;
    }

    DBG("called checkpoint_begin");
    current_ckpt = command_t(rank, command_t::CHECKPOINT, version, name);
    ckpt_filename = current_ckpt.filename(cfg.get("scratch"));
    DBG("CKPT_BEGIN filename is: " << ckpt_filename);
    checkpoint_in_progress = true;

    file_stream.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    file_stream.open(ckpt_filename, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);

    ckpt_check_done = false;
    TIMER_STOP(io_timer_ckpt_begin, " --- CKPT BEGIN TIME --- ");
    return true;
}


void CUDART_CB veloc_client_t::enqueue_write(hipStream_t stream, hipError_t status, void *data) {
    veloc_client_t *th = (veloc_client_t *)data;
    int id = th->gpu_memcpy_queue.front();
    th->gpu_memcpy_queue.pop();
    std::pair<int, region_t> t = std::make_pair(id, th->ckpt_regions[id]);
    DBG("GPU ->Host Memcpy done for " << t.first << " now starting to write to file.");
    th->rem_gpu_cache += std::get<1>(t.second);
    release release_routine = std::get<3>(t.second); 
    // TODO: Call the release_routine function from here....
    std::unique_lock<std::mutex> write_queue_lock(th->write_to_file_mutex);
    th->write_to_file_regions.insert(t);
    th->write_to_file_cv.notify_one();
}

bool veloc_client_t::gpu_to_host_trf() {
    void *ptr; size_t sz;
    while(veloc_client_active){ 
        do {
            std::unique_lock<std::mutex> lock(gpu_memcpy_mutex);
            while (gpu_memcpy_regions.empty()){
                gpu_memcpy_cv.wait(lock, [&](){ return !gpu_memcpy_regions.empty(); }); 
            }
            while (!gpu_memcpy_regions.empty()) {
                // std::unique_lock<std::mutex> ul(gpu_memcpy_done_mutex);
                gpu_memcpy_done = false;
                std::pair<int, region_t> e = *gpu_memcpy_regions.begin();
                // TODO: Unlock 
                DBG("GPU memcpying region "<< e.first);
                char *temp;
                ptr = std::get<0>(e.second);
                sz = std::get<1>(e.second);
                hipHostMalloc((void**)&temp, sz);
                std::get<0>(ckpt_regions[e.first]) = temp;
                hipMemcpyAsync(temp, ptr, sz, hipMemcpyDeviceToHost, veloc_stream);
                gpu_memcpy_queue.push(e.first);
                hipStreamAddCallback(veloc_stream, veloc_client_t::enqueue_write, this, 0);
                gpu_memcpy_regions.erase(e.first);
                temp_host_ptrs.push_back(temp);
            }
        } while (!ckpt_check_done);
        hipStreamSynchronize(veloc_stream);
        // std::unique_lock<std::mutex> ul(gpu_memcpy_done_mutex);
        // gpu_memcpy_done = true;
        // gpu_memcpy_done_cv.notify_one();
    }
    return true;
}

bool veloc_client_t::checkpoint_mem(int mode, std::set<int> &ids) {
    TIMER_START(io_timer_ckpt_mem);
    DBG("Starting checkpoint_mem");
    if (!checkpoint_in_progress) {
        ERROR("must call checkpoint_begin() first");
        return false;
    }
    ckpt_regions.clear();
    if (mode == VELOC_CKPT_ALL)
        ckpt_regions = mem_regions;
    else if (mode == VELOC_CKPT_SOME) {
        for (auto it = ids.begin(); it != ids.end(); it++) {
            auto found = mem_regions.find(*it);
            if (found != mem_regions.end())
                ckpt_regions.insert(*found);
        }
    } else if (mode == VELOC_CKPT_REST) {
        ckpt_regions = mem_regions;
        for (auto it = ids.begin(); it != ids.end(); it++)
            ckpt_regions.erase(*it);
    }
    if (ckpt_regions.size() == 0) {
        ERROR("empty selection of memory regions to checkpoint, please check protection and/or selective checkpointing primitives");
        return false;
    }

    // check_host_enough_memory
    float host_cache = std::stof(current_ckpt.filename(cfg.get("host_cache_size")));
    float gpu_cache = std::stof(current_ckpt.filename(cfg.get("gpu_cache_size")));
    rem_gpu_cache = (1<<30)*gpu_cache;
    DBG("Allowed " << rem_gpu_cache << " GPU cache size.");

    hipPointerAttribute_t attributes;
    void *ptr; size_t sz;    
    unsigned int flags=0;
    size_t free_gpu_mem, total_gpu_mem;

    // TODO: Send this function to the 
    // write_to_file_thread so that the main thread is non-blocking
    // write_headers(ckpt_regions);

    // In this loop, we check for the GPU memory transfer conditions,
    // If gpu_cache_space is available, we either make a copy or start
    // async ckpt of the original dev pointer - We add in gpu_memory_regions 
    // and notify the thread 
    // Else, if gpu_cache_space is unavailable, we perform blocking transfer from
    // the device to host memory.
    for (auto &e : ckpt_regions) {
        ptr = std::get<0>(e.second);
        sz = std::get<1>(e.second);
        flags = std::get<2>(e.second);
        hipPointerGetAttributes (&attributes, ptr);
        if(attributes.type==hipMemoryTypeDevice) {
            // hipMemGetInfo(&free_gpu_mem, &total_gpu_mem);
            DBG("Free cache on GPU is: " << free_gpu_mem << ", and required memory is: " << sz << " for region " << e.first << " addr: " << ptr);
            if(rem_gpu_cache >= sz) {
                char *new_ptr = (char *)ptr;
                if(flags == DEFAULT) {
                    hipMalloc((void**)&new_ptr, sz);
                    rem_gpu_cache -= sz;
                    DBG("Creating a copy on GPU");
                    hipMemcpy(new_ptr, ptr, sz, hipMemcpyDeviceToDevice);
                    temp_dev_ptrs.push_back(new_ptr);
                }
                std::get<0>(e.second) = new_ptr;
                std::unique_lock<std::mutex> lock(gpu_memcpy_mutex);
                gpu_memcpy_regions.insert(e);
                gpu_memcpy_cv.notify_one();
            } else {
                DBG("Not enough free cache on GPU for region " << e.first);
                char *temp;
                hipHostMalloc((void**)&temp, sz);
                hipMemcpy(temp, ptr, sz, hipMemcpyDeviceToHost);
                std::get<0>(e.second) = temp; 
                DBG("Memcpy done for " << e.first << " changing address from: " << ptr << " to: " << std::get<0>(e.second));
                temp_host_ptrs.push_back(temp);  
                std::unique_lock<std::mutex> write_queue_lock(write_to_file_mutex);
                write_to_file_regions.insert(e);
                write_to_file_cv.notify_one();
            }
        } else {
            DBG("Direct host to file transfer for region " << e.first);
            std::unique_lock<std::mutex> write_queue_lock(write_to_file_mutex);
            write_to_file_regions.insert(e);
            write_to_file_cv.notify_one();
        }
    }
    ckpt_check_done = true;

    TIMER_STOP(io_timer_ckpt_mem, " --- CKPT MEM TIME --- ");
    return true;
}

bool veloc_client_t::mem_to_file_write() {    
    while(veloc_client_active) {
        try {        
            do {
                std::unique_lock<std::mutex> lock(write_to_file_mutex);
                while (write_to_file_regions.empty()){
                    write_to_file_cv.wait(lock, [&](){ return !write_to_file_regions.empty(); }); 
                }
                
                while (!write_to_file_regions.empty()) {
                    auto e = write_to_file_regions.begin();
                    int d = std::distance(ckpt_regions.begin(), ckpt_regions.find(e->first));
                    int offset = d + sizeof(size_t) + ckpt_regions.size()*(sizeof(size_t)+sizeof(int));
                    file_stream.seekp(offset);
                    DBG("Starting to write region " << e->first << " at offset: " << offset << " for " << (std::get<0>(e->second)) << " sz: " << std::get<1>(e->second));
                    file_stream.write((char *)&(std::get<0>(e->second)), std::get<1>(e->second));
                    DBG("f.write completed...");
                    write_to_file_regions.erase(e);
                }
            } while (!ckpt_check_done || !gpu_memcpy_done || !write_to_file_regions.empty());
            // Write headers
            file_stream.seekp(0);
            size_t regions_size = ckpt_regions.size();
            file_stream.write((char *)&regions_size, sizeof(size_t));
            for (auto &e : ckpt_regions) {
                file_stream.write((char *)&(e.first), sizeof(int));
                file_stream.write((char *)&(std::get<1>(e.second)), sizeof(size_t));
            }  
            file_stream.close();
            DBG("DONE writing all regions to scratch!");
            // for (char *t : temp_host_ptrs)
            //     hipHostFree(t);
            // for (char *t : temp_dev_ptrs)
            //     hipFree(t);
            // temp_host_ptrs.clear();
            // temp_dev_ptrs.clear();
        } catch (std::ofstream::failure &f) {
            ERROR("cannot write to checkpoint file: " << current_ckpt << ", reason: " << f.what());
            return false;
        }
    }
    return true;
}

bool veloc_client_t::write_headers(regions_t ckpt_regions) {
    std::ofstream f;
    f.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    try {
        f.open(current_ckpt.filename(cfg.get("scratch")), std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
        size_t regions_size = ckpt_regions.size();
        f.write((char *)&regions_size, sizeof(size_t));
        for (auto &e : ckpt_regions) {
            f.write((char *)&(e.first), sizeof(int));
            f.write((char *)&(std::get<1>(e.second)), sizeof(size_t));
        }  
        f.clear();
        f.close();
    } catch (std::ofstream::failure &f) {
        ERROR("cannot write to checkpoint file: " << current_ckpt << ", reason: " << f.what());
        return false;
    }
    return true;
}

bool veloc_client_t::checkpoint_end(bool /*success*/) {
    TIMER_START(io_timer_ckpt_end);
    checkpoint_in_progress = false;
    if (cfg.is_sync()) {
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return modules->notify_command(current_ckpt) == VELOC_SUCCESS;
    }
    else {
        queue->enqueue(current_ckpt);
        TIMER_STOP(io_timer_ckpt_end, " --- CKPT END TIME --- ");
        return true;
    }
}

int veloc_client_t::run_blocking(const command_t &cmd) {
    if (cfg.is_sync())
	return modules->notify_command(cmd);
    else {
	queue->enqueue(cmd);
	return queue->wait_completion();
    }
}

int veloc_client_t::restart_test(const char *name, int needed_version) {
    if (!validate_name(name) || needed_version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }
    int version = run_blocking(command_t(rank, command_t::TEST, needed_version, name));
    DBG(name << ": latest version = " << version);
    if (collective) {
	int min_version;
	MPI_Allreduce(&version, &min_version, 1, MPI_INT, MPI_MIN, comm);
	return min_version;
    } else
	return version;
}

std::string veloc_client_t::route_file(const char *original) {
    char abs_path[PATH_MAX + 1];
    if (original[0] != '/' && getcwd(abs_path, PATH_MAX) != NULL)
	current_ckpt.assign_path(current_ckpt.original, std::string(abs_path) + "/" + std::string(original));
    else
	current_ckpt.assign_path(current_ckpt.original, std::string(original));
    return current_ckpt.filename(cfg.get("scratch"));
}

bool veloc_client_t::restart_begin(const char *name, int version) {
    if (checkpoint_in_progress) {
	INFO("cannot restart while checkpoint in progress");
	return false;
    }
    if (!validate_name(name) || version < 0) {
	ERROR("checkpoint name and/or version incorrect: name can only include [a-zA-Z0-9_] characters, version needs to be non-negative integer");
	return VELOC_FAILURE;
    }

    int result, end_result;
    current_ckpt = command_t(rank, command_t::RESTART, version, name);
    result = run_blocking(current_ckpt);
    if (collective)
	MPI_Allreduce(&result, &end_result, 1, MPI_INT, MPI_LOR, comm);
    else
	end_result = result;
    if (end_result == VELOC_SUCCESS) {
        header_size = 0;
	return true;
    } else
	return false;
}

bool veloc_client_t::read_header() {
    region_info.clear();
    try {
	std::ifstream f;
        size_t expected_size = 0;

	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	size_t no_regions, region_size;
	int id;
	f.read((char *)&no_regions, sizeof(size_t));
	for (unsigned int i = 0; i < no_regions; i++) {
	    f.read((char *)&id, sizeof(int));
	    f.read((char *)&region_size, sizeof(size_t));
	    region_info.insert(std::make_pair(id, region_size));
            expected_size += region_size;
	}
	header_size = f.tellg();
        f.seekg(0, f.end);
        size_t file_size = (size_t)f.tellg() - header_size;
        if (file_size != expected_size)
            throw std::ifstream::failure("file size " + std::to_string(file_size) + " does not match expected size " + std::to_string(expected_size));
    } catch (std::ifstream::failure &e) {
	ERROR("cannot validate header for checkpoint " << current_ckpt << ", reason: " << e.what());
	header_size = 0;
	return false;
    }
    return true;
}

size_t veloc_client_t::recover_size(int id) {
    if (header_size == 0)
        read_header();
    auto it = region_info.find(id);
    if (it == region_info.end())
	return 0;
    else
	return it->second;
}

bool veloc_client_t::recover_mem(int mode, std::set<int> &ids) {
    if (header_size == 0 && !read_header()) {
	ERROR("cannot recover in memory mode if header unavailable or corrupted");
	return false;
    }
    try {
	std::ifstream f;
	f.exceptions(std::ifstream::failbit | std::ifstream::badbit);
	f.open(current_ckpt.filename(cfg.get("scratch")), std::ifstream::in | std::ifstream::binary);
	f.seekg(header_size);
	for (auto &e : region_info) {
	    bool found = ids.find(e.first) != ids.end();
	    if ((mode == VELOC_RECOVER_SOME && !found) || (mode == VELOC_RECOVER_REST && found)) {
		f.seekg(e.second, std::ifstream::cur);
		continue;
	    }
	    if (mem_regions.find(e.first) == mem_regions.end()) {
		ERROR("no protected memory region defined for id " << e.first);
		return false;
	    }
	    if (std::get<1>(mem_regions[e.first]) < e.second) {
		ERROR("protected memory region " << e.first << " is too small ("
		      << std::get<1>(mem_regions[e.first]) << ") to hold required size ("
		      << e.second << ")");
		return false;
	    }
	    f.read((char *)std::get<0>(mem_regions[e.first]), e.second);
	}
    } catch (std::ifstream::failure &e) {
	ERROR("cannot read checkpoint file " << current_ckpt << ", reason: " << e.what());
	return false;
    }
    return true;
}

bool veloc_client_t::restart_end(bool /*success*/) {
    return true;
}
