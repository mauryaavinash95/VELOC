#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include "veloc.h"
#include "heatdis.h"
#define NUM_BLOCKS 512
#define NUM_THREADS 512
static const unsigned int CKPT_FREQ = ITER_TIMES / 3;
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

/*
    This sample application is based on the heat distribution code
    originally developed within the FTI project: github.com/leobago/fti
*/
__device__ long long int error_val;
__global__ void init_data_gpu(int nbLines, int M, int rank, double *h) {
    int j;
    error_val = 0;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    while (idx < nbLines) {
        for (j = 0; j < M; j++) {
            h[(idx*M)+j] = 0;
        }
        idx += blockDim.x;
    }
    if (rank == 0) {
        int start = int(M*0.1), end = ceil(M*0.9);
        idx = threadIdx.x + blockIdx.x*blockDim.x;
        while(idx >= start && idx < end) {
            h[idx] = 100;
            idx += blockDim.x;
        }
    }
}

__global__ void copy_g(int nbLines, int M, double *g, double *h) {
    int j, idx = threadIdx.x + blockIdx.x*blockDim.x;
    while (idx < nbLines) {
        for(j = 0; j < M; j++) {
            h[(idx*M)+j] = g[(idx*M)+j];
        }
        idx += blockDim.x;
    }
}

__global__ void compute(int nbLines, int M, double *g, double *h) {
    int j, i = threadIdx.x + blockIdx.x*blockDim.x;
    double err;
    __shared__ long long int block_err;
    block_err = 0;
    long long int le;
    while (i > 0 && i < (nbLines-1)) {
        for(j = 0; j < M; j++) {
            g[(i*M)+j] = 0.25*(h[((i-1)*M)+j]+h[((i+1)*M)+j]+h[(i*M)+j-1]+h[(i*M)+j+1]);
            err = fabs(g[(i*M)+j] - h[(i*M)+j]);
            le = __double_as_longlong(err);
            // atomicMax(&error_val, le);
            atomicMax(&block_err, le);
        }
        i += blockDim.x;
    }
    atomicMax(&error_val, block_err);
}

__global__ void compute_right(int nbLines, int M, double *g) {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    while(idx < M) {
        g[((nbLines-1)*M)+idx] = g[((nbLines-2)*M)+idx];
        idx += blockDim.x;
    }
}

double doWork(int numprocs, int rank, int M, int nbLines, double *dg, double *dh) {
    MPI_Request req1[2], req2[2];
    MPI_Status status1[2], status2[2];
    long long int *localerror;
    printf("Starting dowork...");
    checkCuda(hipHostMalloc((void**)&localerror, sizeof(long long int)));
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(error_val), &localerror, sizeof(long long int), 0, hipMemcpyHostToDevice));
    double *g, *h;
    checkCuda(hipHostMalloc((void**)&h, sizeof(double *) * M * nbLines));
    checkCuda(hipHostMalloc((void**)&g, sizeof(double *) * M * nbLines));
    copy_g<<<NUM_BLOCKS, NUM_THREADS>>>(nbLines, M, dg, dh);
    hipMemcpy(h, dh, sizeof(double *) * M * nbLines, hipMemcpyDeviceToHost);
    hipMemcpy(g, dg, sizeof(double *) * M * nbLines, hipMemcpyDeviceToHost);
    if (rank > 0) {
        MPI_Isend(g+M, M, MPI_DOUBLE, rank-1, WORKTAG, MPI_COMM_WORLD, &req1[0]);
        MPI_Irecv(h,   M, MPI_DOUBLE, rank-1, WORKTAG, MPI_COMM_WORLD, &req1[1]);
    }
    if (rank < numprocs - 1) {
        MPI_Isend(g+((nbLines-2)*M), M, MPI_DOUBLE, rank+1, WORKTAG, MPI_COMM_WORLD, &req2[0]);
        MPI_Irecv(h+((nbLines-1)*M), M, MPI_DOUBLE, rank+1, WORKTAG, MPI_COMM_WORLD, &req2[1]);
    }
    if (rank > 0) {
        MPI_Waitall(2,req1,status1);
    }
    if (rank < numprocs - 1) {
        MPI_Waitall(2,req2,status2);
    }
    compute<<<NUM_BLOCKS, NUM_THREADS>>>(nbLines, M, dg, dh);
    // hipDeviceSynchronize();
    if (rank == (numprocs-1)) {
        compute_right<<<NUM_BLOCKS, NUM_THREADS>>>(nbLines, M, dg);
    }
    // hipDeviceSynchronize();
    // double t = MPI_Wtime();
    checkCuda(hipMemcpyFromSymbol(localerror, HIP_SYMBOL(error_val), sizeof(long long int), 0, hipMemcpyDeviceToHost));
    // printf("Time taken for MemcpyFromSymbol is: %lf\n", MPI_Wtime()-t);
    free(h);
    free(g);
    return *((double*)localerror);
}

int main(int argc, char *argv[]) {
    int rank, nbProcs, nbLines, i, M, arg;
    double wtime, memSize, localerror, globalerror = 1, inner_time;
    if (argc < 3) {
        printf("Usage: %s <mem_in_mb> <cfg_file>\n", argv[0]);
        exit(1);
    }
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &nbProcs);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    int cudaDevices = 0;
    checkCuda(hipGetDeviceCount(&cudaDevices));
    checkCuda(hipSetDevice(rank%cudaDevices));
    if (sscanf(argv[1], "%d", &arg) != 1) {
        printf("Wrong memory size! See usage\n");
	    exit(3);
    }
    if (VELOC_Init(MPI_COMM_WORLD, argv[2]) != VELOC_SUCCESS) {
        printf("Error initializing VELOC! Aborting...\n");
        exit(2);
    }
	
    M = (int)sqrt((double)(arg * 1024.0 * 1024.0 * nbProcs) / (2 * sizeof(double))); // two matrices needed
    nbLines = (M / nbProcs) + 3;
    
    double *dh, *dg;
    hipMalloc((void**)&dh, sizeof(double *) * M * nbLines);
    hipMalloc((void**)&dg, sizeof(double *) * M * nbLines);
    init_data_gpu<<<NUM_BLOCKS, NUM_THREADS>>>(nbLines, M, rank, dg);
    memSize = M * nbLines * 2 * sizeof(double) / (1024 * 1024);

    if (rank == 0)
	    printf("Local data size is %d x %d = %f MB (%d).\n", M, nbLines, memSize, arg);
    if (rank == 0)
	    printf("Target precision : %f \n", PRECISION);
    if (rank == 0)
	    printf("Maximum number of iterations : %d \n", ITER_TIMES);
    printf("Initing wtime");
    wtime = MPI_Wtime();
    inner_time = MPI_Wtime();
    i = 0;
    printf("Before veloc_mem_protect....");
    VELOC_Mem_protect(0, &i, 1, sizeof(int));
    VELOC_Mem_protect(1, dh, M * nbLines, sizeof(double));
    VELOC_Mem_protect(2, dg, M * nbLines, sizeof(double));
    printf("After veloc_mem_protect....");
    while(i < ITER_TIMES) {
        localerror = doWork(nbProcs, rank, M, nbLines, dg, dh);
        if (((i % ITER_OUT) == 0) && (rank == 0)) {
            printf("Step : %d, error = %f, localerror = %f, time = %lf s\n", i, globalerror, localerror, MPI_Wtime()-inner_time);
            inner_time = MPI_Wtime();
        }
        if ((i % REDUCE) == 0) {
            MPI_Allreduce(&localerror, &globalerror, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
        }
        if (globalerror < PRECISION) {
            printf("Breaking due to error being less: %lf", globalerror);
            break;
        }
        i++;
        if (i % CKPT_FREQ == 0) {
            printf("\n Checkpointing %i\n", i);
            assert(VELOC_Checkpoint("heatdis", i) == VELOC_SUCCESS);
        }
    }
    if (rank == 0)
	    printf("Execution finished in %lf seconds.\n", MPI_Wtime() - wtime);

    hipFree(dh);
    hipFree(dg);
    MPI_Finalize();
    return 0;
}
